#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include "Timer.h"

#if 1
#define OFFSET_PTR(ptr, num_bytes) ((decltype(ptr))((const char*)ptr + num_bytes))
#else
template <typename T>
__device__
T* OFFSET_PTR(T* ptr, int num_bytes)
{
	return (T*)((char*)(ptr) + num_bytes);
}
#endif

__global__
void demosaic(
	const uint16_t* __restrict__ pSrc,
	uchar4* __restrict__ pDst,
	size_t width,
	size_t height,
	size_t src_pitch,
	size_t dst_pitch
	)
{
	//int i;
	//i = gridDim.x;
	//i = blockDim.x;
	//i = blockIdx.x;
	//i = threadIdx.x;

	const size_t y_offset = blockIdx.x * 2;
	const size_t y_loop_count = (height - gridDim.x) / gridDim.x;
	const size_t y_stride = gridDim.x;

	volatile size_t index = threadIdx.x;
	size_t warpIndex = index / warpSize;
	index -= ((warpIndex + 1) >> 1) * warpSize;
	size_t x_offset = 512 * (index / 32u) + (index & 31u) * 2;
	const size_t x_step = 64u;
	const size_t x_loop_count = 8u;
	if (x_offset >= width) {
		return;
	}
	pSrc += x_offset;
	pDst += x_offset;

	pSrc = OFFSET_PTR(pSrc, y_offset * src_pitch);
	pDst = OFFSET_PTR(pDst, y_offset * dst_pitch);
	// �s����
	pSrc = OFFSET_PTR(pSrc, (warpIndex & 1) * src_pitch);
	pDst = OFFSET_PTR(pDst, ((warpIndex & 1) + 1) * dst_pitch) + 1;
	// g r g r g r
	// b g b g b g
	// g r g r g r
	// b g b g b g
	const size_t nShifts = 8;
	const size_t nShifts_p1 = nShifts + 1;
	const size_t nShifts_p2 = nShifts + 2;
	if (warpIndex & 1) {
		for (int y=0; y<y_loop_count; ++y) {
			const uint16_t* pUp = pSrc;
			const uint16_t* pMi = OFFSET_PTR(pSrc, src_pitch);
			const uint16_t* pLo = OFFSET_PTR(pSrc, 2 * src_pitch);
			uchar4* pDst0 = pDst;
			for (int x=0; x<x_loop_count; ++x) {
				uint16_t m1 = pMi[1];
				uint16_t m2 = pMi[2];
				pDst0->x = m1 >> nShifts;
				pDst0->y = (pUp[1] + pMi[0] + m2 + pLo[1] + 2) >> nShifts_p2;
				pDst0->z = (pUp[0] + pUp[2] + pLo[0] + pLo[2] + 2) >> nShifts_p2;
				++pDst0;

				pDst0->x = (m1 + pMi[3] + 1) >> nShifts_p1;
				pDst0->y = m2 >> nShifts;
				pDst0->z = (pUp[2] + pLo[2] + 1) >> nShifts_p1;
				++pDst0;

				pUp += x_step;
				pMi += x_step;
				pLo += x_step;
				pDst0 += x_step - 2;
			}
			pSrc = OFFSET_PTR(pSrc, y_stride * src_pitch);
			pDst = OFFSET_PTR(pDst, y_stride * dst_pitch);
		}
	}else {
		for (int y=0; y<y_loop_count; ++y) {
			const uint16_t* pUp = pSrc;
			const uint16_t* pMi = OFFSET_PTR(pSrc, src_pitch);
			const uint16_t* pLo = OFFSET_PTR(pSrc, 2 * src_pitch);
			uchar4* pDst0 = pDst;
			for (int x=0; x<x_loop_count; ++x) {
				uint32_t r0 = pUp[1] + pLo[1] + 1;
				uint16_t m0 = pMi[0];
				uint16_t m1 = pMi[1];
				uint16_t m2 = pMi[2];
				uint16_t m3 = pMi[3];
				pDst0->x = r0 >> nShifts_p1;
				pDst0->y = m1 >> nShifts;
				pDst0->z = (m0 + m2 + 1) >> nShifts_p1;
				++pDst0;

				pDst0->x = (r0 + pUp[3] + pLo[3] + 1) >> nShifts_p2;
				pDst0->y = (m1 + m3 + 1) >> nShifts_p1;
				pDst0->z = m2 >> nShifts;
				++pDst0;

				pUp += x_step;
				pMi += x_step;
				pLo += x_step;
				pDst0 += x_step - 2;
			}
			pSrc = OFFSET_PTR(pSrc, y_stride * src_pitch);
			pDst = OFFSET_PTR(pDst, y_stride * dst_pitch);
		}
	}

}

void cuda_demosaic_grbg(
	const uint16_t* __restrict pSrc,
	size_t width,
	size_t height,
	uint32_t* __restrict pDst
	)
{
	hipError_t ret;
Timer t;

	uint16_t* d_src;
	uchar4* d_dst;
	size_t d_src_pitch, d_dst_pitch;
	ret = hipHostRegister((void*)pSrc, width*height*2, hipHostRegisterDefault);
	ret = hipHostRegister((void*)pDst, width*height*4, hipHostRegisterDefault);
	ret = hipMallocPitch((void**)&d_src, &d_src_pitch, width*sizeof(uint16_t), height);
	ret = hipMallocPitch((void**)&d_dst, &d_dst_pitch, width*sizeof(uchar4), height);
printf("hipMalloc Elapsed %f\n", t.ElapsedSecond());
t.Start();
	ret = hipMemcpy2D(d_src, d_src_pitch, pSrc, width*2, width*2, height, hipMemcpyHostToDevice);
	hipMemset(d_dst, 0, d_dst_pitch * height);
printf("hipMemcpy Elapsed %f\n", t.ElapsedSecond());
t.Start();

	int numBlocksInAGrid = 64;
	int numThreadsInABlock = 512;

t.Start();
	demosaic<<<numBlocksInAGrid,numThreadsInABlock>>>(d_src, d_dst, width, height, d_src_pitch, d_dst_pitch);
	ret = hipDeviceSynchronize();
printf("demosaic Elapsed %f\n", t.ElapsedSecond());
t.Start();
	ret = hipMemcpy2D(pDst, width*4, d_dst, d_dst_pitch, width*4, height, hipMemcpyDeviceToHost);
printf("hipMemcpy Elapsed %f\n", t.ElapsedSecond());

	ret = hipFree(d_src);
	ret = hipFree(d_dst);
}
